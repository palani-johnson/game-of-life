#include "hip/hip_runtime.h"

/* File:     cuda_game.cu
 *
 * Compile:  nvcc cuda_game.cu game.c -o cuda_game
 * Run:      ./cuda_game [random|rand] [board_width] [board_height] [seed] [fill] [iterations]
 * 
 * Examples: 
 *      Gen 1000x1000 game board with seed 50 and fill 50 and save as game.ppm:
 *           ./cuda_game rand 1000 1000 10 50 600 > game.ppm
 *     
 *      View ppm:
 *           mpv --no-correct-pts --fps=60 game.ppm
 *      
 *      Stream game with pipe into mpv:
 *          ./cuda_game rand 1000 1000 10 50 600 | mpv --no-correct-pts --fps=10 -
 */
extern "C" {
    #include "game_of_life.h"
}

#define cuda_pos(i, j) int i = blockIdx.x * blockDim.x + threadIdx.x;\
    int j = blockIdx.y * blockDim.y + threadIdx.y

#define error_if(check, msg, val) if (check) { fprintf(stderr, msg, val); exit(EXIT_FAILURE); } 

// this number is implementation specific. change accordingly
#define CUDA_NUM 16

// Uses cuda to compute a life buffer.
__global__ void cuda_gen_next_buff(struct GameOfLife *life) {
    cuda_pos(i, j);

    int sum = 0;
    for (int h = j-1; h <= j+1; h++)
        for (int w = i-1; w <= i+1; w++) 
            if (life->buff[game_pos(life, w, h)]) sum++;
    
    int p = game_pos(life, i, j);
    life->next_buff[p] = sum == 3 || (life->buff[p] && sum == 4);
}

// Uses cuda to fill a video buffer.
__global__ void cuda_write_video_buffer(struct GameOfLife *life) {
    cuda_pos(i, j);

    int jh = j * (life->height) * 3;
    int i3 = i*3;

    char b = life->buff[game_pos(life, i, j)] ? 0 : 255;
    for(int c = 0; c < 3; c++) life->vid_buff[i3 + jh + c] = b;
}

struct GameOfLife *cuda_init_life(struct GameOfLife *life, struct GameOfLife *cuda_life_h) {
    memcpy(cuda_life_h, life, sizeof(GameOfLife));

    // allocate the buffers
    hipMalloc(&(cuda_life_h->buff), life->buff_size * sizeof(bool));
    hipMalloc(&(cuda_life_h->next_buff), life->buff_size * sizeof(bool));
    hipMalloc(&(cuda_life_h->vid_buff), life->vid_buff_size * sizeof(char));

    // copy buffer
    hipMemcpy(cuda_life_h->buff, life->buff, life->buff_size, hipMemcpyHostToDevice);

    // make the data struct
    struct GameOfLife *cuda_life_d;
    hipMalloc(&cuda_life_d, sizeof(GameOfLife)); 
    hipMemcpy(cuda_life_d, cuda_life_h, sizeof(GameOfLife), hipMemcpyHostToDevice);

    return cuda_life_d;
}

void ppm_write_from_cuda(struct GameOfLife *life, struct GameOfLife *cuda_life, FILE *f) {
    hipMemcpy(
        life->vid_buff, 
        cuda_life->vid_buff, 
        life->vid_buff_size * sizeof(char), 
        hipMemcpyDeviceToHost
    );

    fprintf(f, "P6\n%d %d 255\n", life->width, life->height);
    fwrite(life->vid_buff, sizeof(char), life->vid_buff_size, f);
    fflush(f);
}

int main(int argc, char** argv) {
    get_env;

    if (argc != 7) {
        fprintf(
            stderr, 
            "usage:  %s [random|rand] [board_width] [board_height] [seed] [fill] [iterations]\n", 
            argv[0]
        );
        exit(EXIT_FAILURE);
    };

    struct GameOfLife life_struct, *life, *cuda_life_d, cuda_life_h_struct, *cuda_life_h;
    life = &life_struct;
    cuda_life_h = &cuda_life_h_struct;

    int iterations = strtol(argv[6], NULL, 10);
    int width = strtol(argv[2], NULL, 10);
    int height = strtol(argv[3], NULL, 10);

    error_if(width < CUDA_NUM, "Width must be >= %d\n", CUDA_NUM)
    error_if(height < CUDA_NUM, "Height must be >= %d\n", CUDA_NUM)
    error_if(width % CUDA_NUM != 0, "Width mod %d must be 0\n", CUDA_NUM)
    error_if(height % CUDA_NUM != 0, "Height mod %d must be 0\n", CUDA_NUM)
    
    dim3 threads(CUDA_NUM, CUDA_NUM);
    dim3 blocks(width/CUDA_NUM, height/CUDA_NUM);

    init_life(
        life,
        argv[1], 
        width, 
        height,
        strtol(argv[4], NULL, 10),
        strtol(argv[5], NULL, 10)
    );

    cuda_life_d = cuda_init_life(life, cuda_life_h);

    cuda_write_video_buffer<<<blocks, threads>>>(cuda_life_d);
    if (DO_IO) ppm_write_from_cuda(life, cuda_life_h, stdout);

    for (int i = 0; i < iterations; i++) {
        cuda_gen_next_buff<<<blocks, threads>>>(cuda_life_d);
        iterate_buff(cuda_life_h);
        hipMemcpy(cuda_life_d, cuda_life_h, sizeof(GameOfLife), hipMemcpyHostToDevice);

        cuda_write_video_buffer<<<blocks, threads>>>(cuda_life_d); 
        if (DO_IO) ppm_write_from_cuda(life, cuda_life_h, stdout);
    }

    return EXIT_SUCCESS;
}