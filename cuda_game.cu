#include "hip/hip_runtime.h"

/* File:     cuda_game.cu
 *
 * Compile:  nvcc cuda_game.cu game.c -o cuda_game
 * Run:      ./game [threads] [random|rand] [board_width] [board_height] [seed] [fill] [iterations]
 * 
 * Examples: 
 *      Gen 1000x1000 game board with seed 50 and fill 50 and save as game.ppm:
 *           ./omp_game 16 rand 1000 1000 10 50 600 > game.ppm
 *     
 *      View ppm:
 *           mpv --no-correct-pts --fps=60 game.ppm
 *      
 *      Stream game with pipe into mpv:
 *          ./omp_game 16 rand 1000 1000 10 50 600 | mpv --no-correct-pts --fps=10 -
 */
#include "game_of_life.h"

__device__ int width;

// Copies data into the extra space of a life buffer so that
// it has the topology of a torus
void make_torus(struct GameOfLife *life) {
    int wm1 = life->width-1;
    int hm1 = life->height-1;
    int bm1 = life->buff_size-1;

    for (int j = 0; j < life->height; j++) {
        life->buff[game_pos(life, life->width, j)] = life->buff[game_pos(life, 0, j)];
        life->buff[game_pos(life, -1, j)] = life->buff[game_pos(life, wm1, j)];
    }

    for (int i = 0; i < life->width; i++) {
        life->buff[game_pos(life, i, life->height)] = life->buff[game_pos(life, i, 0)];
        life->buff[game_pos(life, i, -1)] = life->buff[game_pos(life, i, hm1)];
    }

    life->buff[bm1] = life->buff[game_pos(life, 0, 0)];
    life->buff[0] = life->buff[game_pos(life, wm1, hm1)];
    life->buff[life->width + 1] = life->buff[game_pos(life, 0, hm1)];
    life->buff[bm1 - (life->width + 1)] = life->buff[game_pos(life, wm1, 0)];
}

// Uses cuda to compute a life buffer.
__global__ void cuda_gen_next_buff(struct GameOfLife *life) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    for (int h = j-1; h <= j+1; h++)
        for (int w = i-1; w <= i+1; w++) 
            if (life->buff[game_pos(life, w, h)]) sum++;
    
    int p = game_pos(life, i, j);
    life->next_buff[p] = sum == 3 || (life->buff[p] && sum == 4);
}

// Uses cuda to fill a video buffer.
__global__ void cuda_write_video_buffer(struct GameOfLife *life) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int i3;
    int jh;

    char b = life->buff[game_pos(life, i, j)] ? 0 : 255;
    for(int c = 0; c < 3; c++) life->vid_buff[i3 + jh + c] = b;
}

void cuda_init_life(
    struct GameOfLife *life,
    struct GameOfLife *cuda_life
) {
    // allocate the struct
    hipMalloc((void **)&cuda_life, sizeof(GameOfLife));

    // copy data from struct (pointers will be bad but thats ok for now)
    hipMemcpy(cuda_life, life, sizeof(GameOfLife), hipMemcpyHostToDevice); 

    // allocate the buffers
    hipMalloc((void **)&cuda_life->buff, life->buff_size);
    hipMalloc((void **)&cuda_life->next_buff, life->buff_size);
    hipMalloc((void **)&cuda_life->vid_buff, life->width * life->height * sizeof(char) * 3);

    // copy buffer
    hipMemcpy(cuda_life->buff, life->buff, life->buff_size, hipMemcpyHostToDevice); 
}

int main(int argc, char** argv) {
    if (argc != 7) {
        fprintf(
            stderr, 
            "usage:  %s [random|rand] [board_width] [board_height] [seed] [fill] [iterations]\n", 
            argv[0]
        );
        exit(EXIT_FAILURE);
    };

    struct GameOfLife life_struct, *life, *cuda_life;
    life = &life_struct;

    init_life(
        life,
        argv[1], 
        strtol(argv[2], NULL, 10), 
        strtol(argv[3], NULL, 10),
        strtol(argv[4], NULL, 10),
        strtol(argv[5], NULL, 10)
    );

    cuda_init_life(life, cuda_life);

    // // setup cuda
    // char *cuda_vid_buff;
    // bool *cuda_buff, *cuda_next_buff;

    // cuda_init_life(life, cuda_vid_buff, cuda_buff, cuda_vid_buff)

    // write_video_buffer(life, vid_buff);
    // ppm_write(life, stdout, vid_buff);

    // for (int i = 0; i < strtol(argv[6], NULL, 10); i++) {
    //     make_torus(life);
    //     gen_next_buff(life);
    //     iterate_buff(life);
    //     write_video_buffer(life, vid_buff);  
    //     ppm_write(life, stdout, vid_buff);
    // }

    // free_buffs(life);

    return EXIT_SUCCESS;
}